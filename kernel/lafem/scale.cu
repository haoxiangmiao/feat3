#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/lafem/scale.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_scale(DT_ * r, const DT_ * x, const DT_ s, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        r[idx] = x[idx] * s;
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;

template <typename DT_>
void Scale<Algo::CUDA>::value(DenseVector<Mem::CUDA, DT_> & r, const DenseVector<Mem::CUDA, DT_> & x, const DT_ s)
{
  if (x.size() != r.size())
    throw InternalError("Vector size does not match!");

  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((r.size())/(double)(block.x));

  DT_ * r_gpu(r.elements());
  const DT_ * x_gpu(x.elements());

  FEAST::LAFEM::Intern::cuda_scale<<<grid, block>>>(r_gpu, x_gpu, s, r.size());
}

template void Scale<Algo::CUDA>::value(DenseVector<Mem::CUDA, float> &, const DenseVector<Mem::CUDA, float> &, const float);
template void Scale<Algo::CUDA>::value(DenseVector<Mem::CUDA, double> &, const DenseVector<Mem::CUDA, double> &, const double);

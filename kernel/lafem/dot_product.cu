// includes, FEAST
#include <kernel/lafem/dot_product.hpp>

// includes, CUDA
#include <hipblas.h>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      float cuda_dot_product(const float * x, const float * y, const Index size)
      {
        return hipblasSdot(size, x, 1, y, 1);
      }

      double cuda_dot_product(const double * x, const double * y, const Index size)
      {
        return hipblasDdot(size, x, 1, y, 1);
      }
    }
  }
}

using namespace FEAST;
using namespace FEAST::LAFEM;

template <typename DT_>
DT_ DotProduct<Mem::CUDA, Algo::CUDA>::value(const DenseVector<Mem::CUDA, DT_> & x, const DenseVector<Mem::CUDA, DT_> & y)
{
  if (x.size() != y.size())
    throw InternalError("Vector size does not match!");

  const DT_ * x_gpu(x.elements());
  const DT_ * y_gpu(y.elements());
  cublasInit();
  DT_ result = Intern::cuda_dot_product(x_gpu, y_gpu, x.size());
  cublasShutdown();
  return result;
}

template float DotProduct<Mem::CUDA, Algo::CUDA>::value(const DenseVector<Mem::CUDA, float> &, const DenseVector<Mem::CUDA, float>&);
template double DotProduct<Mem::CUDA, Algo::CUDA>::value(const DenseVector<Mem::CUDA, double> &, const DenseVector<Mem::CUDA, double>&);

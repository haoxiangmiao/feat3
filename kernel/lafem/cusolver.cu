// includes, FEAST
#include <kernel/base_header.hpp>

#ifdef FEAST_HAVE_CUSOLVER
#include <kernel/archs.hpp>
#include <kernel/lafem/cusolver.hpp>
#include <kernel/util/exception.hpp>

#include <hipsolver.h>
#include "hipsparse.h"

using namespace FEAST;
using namespace FEAST::LAFEM;

void CuSolverLU::solve_intern(int n, int nnzA, const double * csrValA, const int * csrRowPtrA, const int * csrColIndA,
    const double * b, double * x)
{
  hipsolverSpHandle_t handle;
  hipsolverSpCreate(&handle);

  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

  int singularity;
  cusolverSpDcsrlsvluHost(handle, n, nnzA, descr, csrValA, csrRowPtrA, csrColIndA, b, 0.0, 1, x, &singularity);

  hipsparseDestroyMatDescr(descr);
  hipsolverSpDestroy(handle);
}

void CuSolverQR::solve_intern(int m, int nnz, const double * csrValA, const int * csrRowPtrA, const int * csrColIndA,
    const double * b, double * x)
{
  hipsolverSpHandle_t handle;
  hipsolverSpCreate(&handle);

  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

  int singularity;
  hipsolverSpDcsrlsvqr(handle, m, nnz, descr, csrValA, csrRowPtrA, csrColIndA, b, 0.0, 1, x, &singularity);

  hipsparseDestroyMatDescr(descr);
  hipsolverSpDestroy(handle);
}
#endif // FEAST_HAVE_CUSOLVER

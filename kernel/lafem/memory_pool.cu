#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/lafem/memory_pool.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_set_memory(DT_ * ptr, const DT_ val, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        ptr[idx] = val;
      }

      __global__ void cuda_generate_hash(char * cd, const Index bytes, void * resultv)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx != 0)
          return;

        unsigned long * result = (unsigned long *)resultv;
        unsigned long t(0);
        for (Index i(0) ; i < bytes ; ++i)
        {
          t += (cd[i] * i) % bytes;
        }
        t = t % bytes;
        result[0] = t;
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;

MemoryPool<Mem::CUDA>::MemoryPool()
{
}

MemoryPool<Mem::CUDA>::~MemoryPool()
{
  if (_pool.size() > 0)
    throw InternalError("Memory Pool<GPU> still contains memory chunks!");
}

template <typename DT_>
DT_ * MemoryPool<Mem::CUDA>::allocate_memory(Index bytes)
{
  DT_ * memory(NULL);
  if (hipErrorOutOfMemory == hipMalloc((void**)&memory, bytes))
    throw InternalError("MemoryPool<GPU> cuda allocation error (hipErrorOutOfMemory)");
  if (memory == NULL)
    throw InternalError("MemoryPool<GPU> allocation error (null pointer returned)");
  Intern::MemoryInfo mi;
  mi.counter = 1;
  mi.size = bytes;
  _pool.insert(std::pair<void*, Intern::MemoryInfo>(memory, mi));
  return memory;
}

void MemoryPool<Mem::CUDA>::increase_memory(void * address)
{
  std::map<void*, Intern::MemoryInfo>::iterator it(_pool.find(address));
  if (it == _pool.end())
    throw InternalError("MemoryPool<GPU>::increase_memory: Memory address not found!");
  else
  {
    it->second.counter = it->second.counter + 1;
  }
}

void MemoryPool<Mem::CUDA>::release_memory(void * address)
{
  std::map<void*, Intern::MemoryInfo>::iterator it(_pool.find(address));
  if (it == _pool.end())
    throw InternalError("MemoryPool<GPU>::relase_memory: Memory address not found!");
  else
  {
    if(it->second.counter == 1)
    {
      hipFree(address);
      _pool.erase(it);
    }
    else
    {
      it->second.counter = it->second.counter - 1;
    }
  }
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::download(void * dest, void * src, Index bytes)
{
  hipMemcpy(dest, src, bytes, hipMemcpyDeviceToHost);
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::upload(void * dest, void * src, Index bytes)
{
  hipMemcpy(dest, src, bytes, hipMemcpyHostToDevice);
}

template <typename DT_>
DT_ MemoryPool<Mem::CUDA>::get_element(const DT_ * data, Index index)
{
  const void * src(data + index);
  DT_ value;
  hipMemcpy(&value, src, sizeof(DT_), hipMemcpyDeviceToHost);
  return value;
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::set_memory(DT_ * address, const DT_ val, const Index count)
{
  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((count)/(double)(block.x));
  FEAST::LAFEM::Intern::cuda_set_memory<<<grid, block>>>(address, val, count);
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::copy(void * dest, const void * src, const Index bytes)
{
  if (dest == src)
    return;

  hipMemcpy(dest, src, bytes, hipMemcpyDeviceToDevice);
}

unsigned long MemoryPool<Mem::CUDA>::generate_hash(void * data, Index bytes)
{
  dim3 grid(1,1,1);
  dim3 block(128,1,1);
  unsigned long result(0);
  void * result_gpu(0);
  hipMalloc((void**)&result_gpu, sizeof(unsigned long));
  char * datac((char *)data);
  FEAST::LAFEM::Intern::cuda_generate_hash<<<grid, block>>>(datac, bytes, result_gpu);
  hipMemcpy(&result, result_gpu, sizeof(unsigned long), hipMemcpyDeviceToHost);
  hipFree(result_gpu);
  return result;
}

template float * MemoryPool<Mem::CUDA>::allocate_memory<float>(Index bytes);
template double * MemoryPool<Mem::CUDA>::allocate_memory<double>(Index bytes);
template unsigned long * MemoryPool<Mem::CUDA>::allocate_memory<unsigned long>(Index bytes);

template void MemoryPool<Mem::CUDA>::download<float>(void * dest, void * src, Index bytes);
template void MemoryPool<Mem::CUDA>::download<double>(void * dest, void * src, Index bytes);
template void MemoryPool<Mem::CUDA>::download<unsigned long>(void * dest, void * src, Index bytes);

template void MemoryPool<Mem::CUDA>::upload<float>(void * dest, void * src, Index bytes);
template void MemoryPool<Mem::CUDA>::upload<double>(void * dest, void * src, Index bytes);
template void MemoryPool<Mem::CUDA>::upload<unsigned long>(void * dest, void * src, Index bytes);

template float MemoryPool<Mem::CUDA>::get_element(const float * data, Index index);
template double MemoryPool<Mem::CUDA>::get_element(const double * data, Index index);
template Index MemoryPool<Mem::CUDA>::get_element(const Index * data, Index index);

template void MemoryPool<Mem::CUDA>::set_memory(float * address , const float val, const Index count);
template void MemoryPool<Mem::CUDA>::set_memory(double * address , const double val, const Index count);
template void MemoryPool<Mem::CUDA>::set_memory(Index * address , const Index val, const Index count);

template void MemoryPool<Mem::CUDA>::copy<float>(void * dest, const void * src, const Index Bytes);
template void MemoryPool<Mem::CUDA>::copy<double>(void * dest, const void * src, const Index Bytes);
template void MemoryPool<Mem::CUDA>::copy<unsigned long>(void * dest, const void * src, const Index Bytes);

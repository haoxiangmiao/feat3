#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/lafem/memory_pool.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_set_memory(DT_ * ptr, const DT_ val, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        ptr[idx] = val;
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;

MemoryPool<Mem::CUDA>::MemoryPool()
{
}

MemoryPool<Mem::CUDA>::~MemoryPool()
{
  if (_pool.size() > 0)
    throw InternalError("Memory Pool<GPU> still contains memory chunks!");
}

template <typename DT_>
DT_ * MemoryPool<Mem::CUDA>::allocate_memory(const Index count)
{
  DT_ * memory(NULL);
  if (hipErrorOutOfMemory == hipMalloc((void**)&memory, count * sizeof(DT_)))
    throw InternalError("MemoryPool<GPU> cuda allocation error (hipErrorOutOfMemory)");
  if (memory == NULL)
    throw InternalError("MemoryPool<GPU> allocation error (null pointer returned)");
  Intern::MemoryInfo mi;
  mi.counter = 1;
  mi.size = count * sizeof(DT_);
  _pool.insert(std::pair<void*, Intern::MemoryInfo>(memory, mi));
  return memory;
}

void MemoryPool<Mem::CUDA>::increase_memory(void * address)
{
  std::map<void*, Intern::MemoryInfo>::iterator it(_pool.find(address));
  if (it == _pool.end())
    throw InternalError("MemoryPool<GPU>::increase_memory: Memory address not found!");
  else
  {
    it->second.counter = it->second.counter + 1;
  }
}

void MemoryPool<Mem::CUDA>::release_memory(void * address)
{
  std::map<void*, Intern::MemoryInfo>::iterator it(_pool.find(address));
  if (it == _pool.end())
    throw InternalError("MemoryPool<GPU>::relase_memory: Memory address not found!");
  else
  {
    if(it->second.counter == 1)
    {
      hipFree(address);
      _pool.erase(it);
    }
    else
    {
      it->second.counter = it->second.counter - 1;
    }
  }
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::download(DT_ * dest, DT_ * src, const Index count)
{
  hipMemcpy(dest, src, count * sizeof(DT_), hipMemcpyDeviceToHost);
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::upload(DT_ * dest, DT_ * src, const Index count)
{
  hipMemcpy(dest, src, count * sizeof(DT_), hipMemcpyHostToDevice);
}

template <typename DT_>
DT_ MemoryPool<Mem::CUDA>::get_element(const DT_ * data, const Index index)
{
  const void * src(data + index);
  DT_ value;
  hipMemcpy(&value, src, sizeof(DT_), hipMemcpyDeviceToHost);
  return value;
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::set_memory(DT_ * address, const DT_ val, const Index count)
{
  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((count)/(double)(block.x));
  FEAST::LAFEM::Intern::cuda_set_memory<<<grid, block>>>(address, val, count);
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::copy(DT_ * dest, const DT_ * src, const Index count)
{
  if (dest == src)
    return;

  hipMemcpy(dest, src, count * sizeof(DT_), hipMemcpyDeviceToDevice);
}

template float * MemoryPool<Mem::CUDA>::allocate_memory<float>(const Index);
template double * MemoryPool<Mem::CUDA>::allocate_memory<double>(const Index);
template Index * MemoryPool<Mem::CUDA>::allocate_memory<Index>(const Index);

template void MemoryPool<Mem::CUDA>::download<float>(float *, float *, const Index);
template void MemoryPool<Mem::CUDA>::download<double>(double *, double *, const Index);
template void MemoryPool<Mem::CUDA>::download<Index>(Index *, Index *, const Index);

template void MemoryPool<Mem::CUDA>::upload<float>(flaot *, float *, const Index);
template void MemoryPool<Mem::CUDA>::upload<double>(double *, double * const Index);
template void MemoryPool<Mem::CUDA>::upload<Index>(Index *, Index *, const Index);

template float MemoryPool<Mem::CUDA>::get_element(const float *, const Index);
template double MemoryPool<Mem::CUDA>::get_element(const double *, const Index);
template Index MemoryPool<Mem::CUDA>::get_element(const Index *, const Index);

template void MemoryPool<Mem::CUDA>::set_memory(float * , const float, const const Index);
template void MemoryPool<Mem::CUDA>::set_memory(double * , const double, const const Index);
template void MemoryPool<Mem::CUDA>::set_memory(Index * , const Index, const Index);

template void MemoryPool<Mem::CUDA>::copy<float>(float *, const float *, const Index);
template void MemoryPool<Mem::CUDA>::copy<double>(double *, const double *, const Index);
template void MemoryPool<Mem::CUDA>::copy<Index>(Index *, const Index *, const Index);

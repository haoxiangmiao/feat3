#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/lafem/memory_pool.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_set_memory(DT_ * ptr, const DT_ val, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        ptr[idx] = val;
      }

      __global__ void cuda_generate_hash(char * cd, const Index bytes, unsigned long * result)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx != 0)
          return;

        unsigned long t(0);
        for (Index i(0) ; i < bytes ; ++i)
        {
          t += (cd[i] * i) % bytes;
        }
        t = t % bytes;
        result[0] = t;
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;

MemoryPool<Mem::CUDA>::MemoryPool()
{
}

MemoryPool<Mem::CUDA>::~MemoryPool()
{
  if (_pool.size() > 0)
    throw InternalError("Memory Pool<GPU> still contains memory chunks!");
}

void * MemoryPool<Mem::CUDA>::allocate_memory(Index bytes)
{
  void * memory(NULL);
  if (hipErrorOutOfMemory == hipMalloc((void**)&memory, bytes))
    throw InternalError("MemoryPool<GPU> cuda allocation error");
  if (memory == NULL)
    throw InternalError("MemoryPool<GPU> allocation error!");
  Intern::MemoryInfo mi;
  mi.counter = 1;
  mi.size = bytes;
  _pool.insert(std::pair<void*, Intern::MemoryInfo>(memory, mi));
  return memory;
}

void MemoryPool<Mem::CUDA>::increase_memory(void * address)
{
  std::map<void*, Intern::MemoryInfo>::iterator it(_pool.find(address));
  if (it == _pool.end())
    throw InternalError("MemoryPool<GPU>::increase_memory: Memory address not found!");
  else
  {
    it->second.counter = it->second.counter + 1;
  }
}

void MemoryPool<Mem::CUDA>::release_memory(void * address)
{
  std::map<void*, Intern::MemoryInfo>::iterator it(_pool.find(address));
  if (it == _pool.end())
    throw InternalError("MemoryPool<GPU>::relase_memory: Memory address not found!");
  else
  {
    if(it->second.counter == 1)
    {
      hipFree(address);
      _pool.erase(it);
    }
    else
    {
      it->second.counter = it->second.counter - 1;
    }
  }
}

void MemoryPool<Mem::CUDA>::download(void * dest, void * src, Index bytes)
{
  hipMemcpy(dest, src, bytes, hipMemcpyDeviceToHost);
}

void MemoryPool<Mem::CUDA>::upload(void * dest, void * src, Index bytes)
{
  hipMemcpy(dest, src, bytes, hipMemcpyHostToDevice);
}

template <typename DT_>
DT_ MemoryPool<Mem::CUDA>::get_element(const DT_ * data, Index index)
{
  const void * src(data + index);
  DT_ value;
  hipMemcpy(&value, src, sizeof(DT_), hipMemcpyDeviceToHost);
  return value;
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::modify_element(DT_ * data, Index index, DT_ value)
{
  void * dest(data + index);
  hipMemcpy(dest, &value, sizeof(DT_), hipMemcpyHostToDevice);
}

template <typename DT_>
void MemoryPool<Mem::CUDA>::set_memory(DT_ * address, const DT_ val, const Index count)
{
  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((count)/(double)(block.x));
  FEAST::LAFEM::Intern::cuda_set_memory<<<grid, block>>>(address, val, count);
}

void MemoryPool<Mem::CUDA>::copy(void * dest, const void * src, const Index bytes)
{
  if (dest == src)
    return;

  hipMemcpy(dest, src, bytes, hipMemcpyDeviceToDevice);
}

unsigned long MemoryPool<Mem::CUDA>::generate_hash(void * data, Index bytes)
{
  dim3 grid(1,1,1);
  dim3 block(128,1,1);
  unsigned long result(0);
  unsigned long * result_gpu;
  hipMalloc((void**)&result_gpu, sizeof(unsigned long));
  char * datac((char *)data);
  FEAST::LAFEM::Intern::cuda_generate_hash<<<grid, block>>>(datac, bytes, result_gpu);
  hipMemcpy(&result, result_gpu, sizeof(unsigned long), hipMemcpyDeviceToHost);
  hipFree(result_gpu);
  return result;
}

template float MemoryPool<Mem::CUDA>::get_element(const float * data, Index index);
template double MemoryPool<Mem::CUDA>::get_element(const double * data, Index index);
template Index MemoryPool<Mem::CUDA>::get_element(const Index * data, Index index);
template void MemoryPool<Mem::CUDA>::modify_element(float * data, Index index, float value);
template void MemoryPool<Mem::CUDA>::modify_element(double * data, Index index, double value);
template void MemoryPool<Mem::CUDA>::modify_element(Index * data, Index index, Index value);
template void MemoryPool<Mem::CUDA>::set_memory(float * address , const float val, const Index count);
template void MemoryPool<Mem::CUDA>::set_memory(double * address , const double val, const Index count);
template void MemoryPool<Mem::CUDA>::set_memory(Index * address , const Index val, const Index count);

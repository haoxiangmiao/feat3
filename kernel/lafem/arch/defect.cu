#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/defect.hpp>
#include <kernel/lafem/arch/difference.hpp>
#include <kernel/lafem/arch/product_matvec.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

#include "hipsparse.h"

namespace FEAST
{
  namespace Util
  {
    namespace Intern
    {
      extern hipsparseHandle_t cusparse_handle;
    }
  }
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_defect_csr(DT_ * r, const DT_ * rhs, const DT_ * b, const DT_ * val, const unsigned long * col_ind,
                                      const unsigned long * row_ptr, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;

        DT_ sum(0);
        const Index end(row_ptr[idx + 1]);
        for (Index i(row_ptr[idx]) ; i < end ; ++i)
        {
          sum += val[i] * b[col_ind[i]];
        }
        r[idx] = rhs[idx] - sum;
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_defect_ell(DT_ * r, const DT_ * rhs, const DT_ * x, const DT_ * val, const IT_ * col_ind,
                                      const IT_ * cs, const IT_ * cl, const Index rows, const Index C)
      {
        const Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;


        DT_ sum(0);
        const Index chunk(idx / C);
        const Index local_row(idx % C);
        const Index chunk_end(cs[chunk+1]);

        for (Index pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol+=C)
        {
          sum += val[pcol] * x[col_ind[pcol]];
        }
        r[idx] = rhs[idx] - sum;

      }

      template <typename DT_, typename IT_>
      __global__ void cuda_defect_banded(DT_ * r, const DT_ * rhs, const DT_ * x, const DT_ * val, const IT_ * offsets, const Index num_of_offsets, const Index rows, const Index columns)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;

        const Index k1(rows - 1);
        const Index k2(rows + columns - 1);

        Index start(0);

        while (k1 > offsets[start] + idx)
        {
          ++start;
        }

        Index end(start);

        while (end < num_of_offsets && idx + offsets[end] < k2)
        {
          ++end;
        }

        DT_ sum(DT_(0.0));
        for (Index diag(start); diag < end; ++diag)
        {
          sum += val[rows * diag + idx] * x[idx + offsets[diag] - rows + 1];
        }
        r[idx] = rhs[idx] - sum;
      }

      void cusparse_defect_csr(hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const float * alpha, const hipsparseMatDescr_t descrA,
                                       const float * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       const float * x, const float * beta, float * y)
      {
        hipsparseScsrmv(Util::Intern::cusparse_handle, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, x, beta, y);
      }

      void cusparse_defect_csr(hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const double * alpha, const hipsparseMatDescr_t descrA,
                                       const double * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       const double * x, const double * beta, double * y)
      {
        hipsparseDcsrmv(Util::Intern::cusparse_handle, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, x, beta, y);
      }

      void cusparse_defect_csrb(hipsparseDirection_t dir, hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const float * alpha, const hipsparseMatDescr_t descrA,
                                       const float * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       int block_dim,
                                       const float * x, const float * beta, float * y)
      {
        hipsparseSbsrmv(Util::Intern::cusparse_handle, dir, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, block_dim, x, beta, y);
      }

      void cusparse_defect_csrb(hipsparseDirection_t dir, hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const double * alpha, const hipsparseMatDescr_t descrA,
                                       const double * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       int block_dim,
                                       const double * x, const double * beta, double * y)
      {
        hipsparseDbsrmv(Util::Intern::cusparse_handle, dir, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, block_dim, x, beta, y);
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
void Defect<Mem::CUDA>::csr(DT_ * r, const DT_ * const rhs, const DT_ * const val, const unsigned long * const col_ind, const unsigned long * const row_ptr, const DT_ * const x, const Index rows, const Index columns, const Index used_elements)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_spmv;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_defect_csr<<<grid, block>>>(r, rhs, x, val, col_ind, row_ptr, rows);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Defect<Mem::CUDA>::csr(float *, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const float * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA>::csr(double *, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const double * const, const Index, const Index, const Index);

template <typename DT_>
void Defect<Mem::CUDA>::csr(DT_ * r, const DT_ * const rhs, const DT_ * const val, const unsigned int * const col_ind, const unsigned int * const row_ptr, const DT_ * const x, const Index rows, const Index columns, const Index used_elements)
{
  const DT_ a(-1.);
  if (r == rhs)
  {
    hipsparseMatDescr_t descr=0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    DT_ one(1);
    FEAST::LAFEM::Intern::cusparse_defect_csr(HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind, x, &one, r);

    hipsparseDestroyMatDescr(descr);
  }
  else
  {
    hipMemcpy(r, rhs, rows * sizeof(DT_), hipMemcpyDeviceToDevice);

    hipsparseMatDescr_t descr=0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    DT_ one(1);
    FEAST::LAFEM::Intern::cusparse_defect_csr(HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind, x, &one, r);

    hipsparseDestroyMatDescr(descr);
  }

#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Defect<Mem::CUDA>::csr(float *, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const float * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA>::csr(double *, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const double * const, const Index, const Index, const Index);

template <typename DT_>
void Defect<Mem::CUDA>::csrb_intern(DT_ * r, const DT_ * const rhs, const DT_ * const val, const unsigned int * const col_ind, const unsigned int * const row_ptr, const DT_ * const x, const Index rows, const Index columns, const Index used_elements, const int blocksize)
{
  const DT_ a(-1.);
  if (r == rhs)
  {
    hipsparseMatDescr_t descr=0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    DT_ one(1);
    FEAST::LAFEM::Intern::cusparse_defect_csrb(HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind,
        blocksize, x, &one, r);

    hipsparseDestroyMatDescr(descr);
  }
  else
  {
    hipMemcpy(r, rhs, rows * blocksize * sizeof(DT_), hipMemcpyDeviceToDevice);

    hipsparseMatDescr_t descr=0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    DT_ one(1);
    FEAST::LAFEM::Intern::cusparse_defect_csrb(HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind,
        blocksize, x, &one, r);

    hipsparseDestroyMatDescr(descr);
  }

#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Defect<Mem::CUDA>::csrb_intern(float *, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const float * const, const Index, const Index, const Index, const int);
template void Defect<Mem::CUDA>::csrb_intern(double *, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const double * const, const Index, const Index, const Index, const int);


template <typename DT_, typename IT_>
void Defect<Mem::CUDA>::ell(DT_ * r, const DT_ * const rhs, const DT_ * const val, const IT_ * const col_ind, const IT_ * const cs, const IT_ * const cl, const DT_ * const x, const Index C, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_spmv;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_defect_ell<<<grid, block>>>(r, rhs, x, val, col_ind, cs, cl, rows, C);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Defect<Mem::CUDA>::ell(float *, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const unsigned long * const, const float * const, const Index, const Index);
template void Defect<Mem::CUDA>::ell(double *, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const unsigned long * const, const double * const, const Index, const Index);
template void Defect<Mem::CUDA>::ell(float *, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const unsigned int * const, const float * const, const Index, const Index);
template void Defect<Mem::CUDA>::ell(double *, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const unsigned int * const, const double * const, const Index, const Index);

template <typename DT_, typename IT_>
void Defect<Mem::CUDA>::banded(DT_ * r, const DT_ * const rhs, const DT_ * const val, const IT_ * const offsets, const DT_ * const x, const Index num_of_offsets, const Index rows, const Index columns)
{
  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_defect_banded<<<grid, block>>>(r, rhs, x, val, offsets, num_of_offsets, rows, columns);
}
template void Defect<Mem::CUDA>::banded(float *, const float *, const float * const, const unsigned long * const, const float * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA>::banded(double *, const double *, const double * const, const unsigned long * const, const double * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA>::banded(float *, const float *, const float * const, const unsigned int * const, const float * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA>::banded(double *, const double *, const double * const, const unsigned int * const, const double * const, const Index, const Index, const Index);

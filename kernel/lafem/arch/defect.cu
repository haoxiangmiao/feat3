#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/defect.hpp>
#include <kernel/lafem/arch/difference.hpp>
#include <kernel/lafem/arch/product_matvec.hpp>
#include <kernel/util/exception.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_defect_csr(DT_ * r, const DT_ * rhs, const DT_ * b, const DT_ * val, const unsigned long * col_ind,
                                      const unsigned long * row_ptr, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;

        DT_ sum(0);
        const Index end(row_ptr[idx + 1]);
        for (Index i(row_ptr[idx]) ; i < end ; ++i)
        {
          sum += val[i] * b[col_ind[i]];
        }
        r[idx] = rhs[idx] - sum;
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_defect_ell(DT_ * r, const DT_ * rhs, const DT_ * x, const DT_ * val, const IT_ * col_ind,
                                      const IT_ * cs, const IT_ * cl, const Index rows, const Index C)
      {
        const Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;


        DT_ sum(0);
        const Index chunk(idx / C);
        const Index local_row(idx % C);
        const Index chunk_end(cs[chunk+1]);

        for (Index pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol+=C)
        {
          sum += val[pcol] * x[col_ind[pcol]];
        }
        r[idx] = rhs[idx] - sum;

      }

      template <typename DT_, typename IT_>
      __global__ void cuda_defect_banded(DT_ * r, const DT_ * rhs, const DT_ * x, const DT_ * val, const IT_ * offsets, const Index num_of_offsets, const Index rows, const Index columns)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;

        const Index k1(rows - 1);
        const Index k2(rows + columns - 1);

        Index start(0);

        while (k1 > offsets[start] + idx)
        {
          ++start;
        }

        Index end(start);

        while (end < num_of_offsets && idx + offsets[end] < k2)
        {
          ++end;
        }

        DT_ sum(DT_(0.0));
        for (Index diag(start); diag < end; ++diag)
        {
          sum += val[rows * diag + idx] * x[idx + offsets[diag] - rows + 1];
        }
        r[idx] = rhs[idx] - sum;
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
void Defect<Mem::CUDA, Algo::CUDA>::csr(DT_ * r, const DT_ * const rhs, const DT_ * const val, const unsigned long * const col_ind, const unsigned long * const row_ptr, const DT_ * const x, const Index rows, const Index columns, const Index used_elements)
{
  Index blocksize(256);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_defect_csr<<<grid, block>>>(r, rhs, x, val, col_ind, row_ptr, rows);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Defect<Mem::CUDA, Algo::CUDA>::csr(float *, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const float * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA, Algo::CUDA>::csr(double *, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const double * const, const Index, const Index, const Index);

template <typename DT_>
void Defect<Mem::CUDA, Algo::CUDA>::csr(DT_ * r, const DT_ * const rhs, const DT_ * const val, const unsigned int * const col_ind, const unsigned int * const row_ptr, const DT_ * const x, const Index rows, const Index columns, const Index used_elements)
{
  FEAST::LAFEM::Arch::ProductMatVec<Mem::CUDA, Algo::CUDA>::csr(r, val, col_ind, row_ptr, x, rows, columns, used_elements);
  FEAST::LAFEM::Arch::Difference<Mem::CUDA, Algo::CUDA>::value(r, rhs, r, rows);
}
template void Defect<Mem::CUDA, Algo::CUDA>::csr(float *, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const float * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA, Algo::CUDA>::csr(double *, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const double * const, const Index, const Index, const Index);


template <typename DT_, typename IT_>
void Defect<Mem::CUDA, Algo::CUDA>::ell(DT_ * r, const DT_ * const rhs, const DT_ * const val, const IT_ * const col_ind, const IT_ * const cs, const IT_ * const cl, const DT_ * const x, const Index C, const Index rows)
{
  Index blocksize(256);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_defect_ell<<<grid, block>>>(r, rhs, x, val, col_ind, cs, cl, rows, C);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Defect<Mem::CUDA, Algo::CUDA>::ell(float *, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const unsigned long * const, const float * const, const Index, const Index);
template void Defect<Mem::CUDA, Algo::CUDA>::ell(double *, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const unsigned long * const, const double * const, const Index, const Index);
template void Defect<Mem::CUDA, Algo::CUDA>::ell(float *, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const unsigned int * const, const float * const, const Index, const Index);
template void Defect<Mem::CUDA, Algo::CUDA>::ell(double *, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const unsigned int * const, const double * const, const Index, const Index);

template <typename DT_, typename IT_>
void Defect<Mem::CUDA, Algo::CUDA>::banded(DT_ * r, const DT_ * const rhs, const DT_ * const val, const IT_ * const offsets, const DT_ * const x, const Index num_of_offsets, const Index rows, const Index columns)
{
  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_defect_banded<<<grid, block>>>(r, rhs, x, val, offsets, num_of_offsets, rows, columns);
}
template void Defect<Mem::CUDA, Algo::CUDA>::banded(float *, const float *, const float * const, const unsigned long * const, const float * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA, Algo::CUDA>::banded(double *, const double *, const double * const, const unsigned long * const, const double * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA, Algo::CUDA>::banded(float *, const float *, const float * const, const unsigned int * const, const float * const, const Index, const Index, const Index);
template void Defect<Mem::CUDA, Algo::CUDA>::banded(double *, const double *, const double * const, const unsigned int * const, const double * const, const Index, const Index, const Index);

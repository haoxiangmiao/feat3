#include "hip/hip_runtime.h"
// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/axpy.hpp>
#include <kernel/lafem/arch/component_product.hpp>
#include <kernel/lafem/arch/product_matvec.hpp>
#include <kernel/lafem/arch/scale.hpp>
#include <kernel/lafem/arch/sum.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

#include "hipsparse.h"

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_axpy(DT_ * r, const DT_ a, const DT_ * x, const DT_ * y, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        r[idx] = a * x[idx] + y[idx];
      }

      template <typename DT_>
      __global__ void cuda_axpy_mv_csr(DT_ * r, const DT_ a, const DT_ * x, const DT_ * y, const DT_ * val,
          const unsigned long * col_ind, const unsigned long * row_ptr, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;

        DT_ sum(0);
        const Index end(row_ptr[idx + 1]);
        for (Index i(row_ptr[idx]) ; i < end ; ++i)
        {
          sum += val[i] * x[col_ind[i]];
        }
        r[idx] = (sum * a) + y[idx];
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_axpy_mv_ell(DT_ * r, const DT_ a, const DT_ * x, const DT_ * y, const DT_ * val, const IT_ * col_ind,
                                       const IT_ * cs, const IT_ * cl, const Index rows, const Index C)
      {
        const Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;


        DT_ sum(0);
        const Index chunk(idx / C);
        const Index local_row(idx % C);
        const Index chunk_end(cs[chunk+1]);

        for (Index pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol+=C)
        {
          sum += val[pcol] * x[col_ind[pcol]];
        }
        r[idx] = sum * a + y[idx];

      }

      template <typename DT_, typename IT_>
      __global__ void cuda_axpy_banded(DT_ * r, const DT_ alpha, const DT_ * x, const DT_ * y, const DT_ * val, const IT_ * offsets, const Index num_of_offsets, const Index rows, const Index columns)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;

        const Index k1(rows - 1);
        const Index k2(rows + columns - 1);

        Index start(0);

        while (k1 > offsets[start] + idx)
        {
          ++start;
        }

        Index end(start);

        while (end < num_of_offsets && idx + offsets[end] < k2)
        {
          ++end;
        }

        DT_ sum(DT_(0.0));
        for (Index diag(start); diag < end; ++diag)
        {
          sum += val[rows * diag + idx] * x[idx + offsets[diag] - rows + 1];
        }
        r[idx] = (sum*alpha) + y[idx];
      }

      void cusparse_axpy_csr(hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const float * alpha, const hipsparseMatDescr_t descrA,
                                       const float * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       const float * x, const float * beta, float * y)
      {
        hipsparseStatus_t status;
        status = hipsparseScsrmv(Util::Intern::cusparse_handle, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, x, beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrmv failed with status code: " + stringify(status));
      }

      void cusparse_axpy_csr(hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const double * alpha, const hipsparseMatDescr_t descrA,
                                       const double * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       const double * x, const double * beta, double * y)
      {
        hipsparseStatus_t status;
        status = hipsparseDcsrmv(Util::Intern::cusparse_handle, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, x, beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrmv failed with status code: " + stringify(status));
      }

      void cusparse_axpy_csrb(hipsparseDirection_t dir, hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const float * alpha, const hipsparseMatDescr_t descrA,
                                       const float * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       int block_dim,
                                       const float * x, const float * beta, float * y)
      {
        hipsparseStatus_t status;
        status = hipsparseSbsrmv(Util::Intern::cusparse_handle, dir, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, block_dim, x, beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrmv failed with status code: " + stringify(status));
      }

      void cusparse_axpy_csrb(hipsparseDirection_t dir, hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const double * alpha, const hipsparseMatDescr_t descrA,
                                       const double * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       int block_dim,
                                       const double * x, const double * beta, double * y)
      {
        hipsparseStatus_t status;
        status = hipsparseDbsrmv(Util::Intern::cusparse_handle, dir, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, block_dim, x, beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrmv failed with status code: " + stringify(status));
      }

      void cublas_axpy_dense(hipblasOperation_t trans,
                                       int m, int n,
                                       const float * alpha,
                                       const float * val,
                                       const float * x, const float * beta, float * y)
      {
        hipblasStatus_t status;
        status = hipblasSgemv(Util::Intern::cublas_handle, trans, n, m, alpha, val, n, x, 1, beta, y, 1);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasSgemv failed with status code: " + stringify(status));
      }

      void cublas_axpy_dense(hipblasOperation_t trans,
                                       int m, int n,
                                       const double * alpha,
                                       const double * val,
                                       const double * x, const double * beta, double * y)
      {
        hipblasStatus_t status;
        status = hipblasDgemv(Util::Intern::cublas_handle, trans, n, m, alpha, val, n, x, 1, beta, y, 1);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasDgemv failed with status code: " + stringify(status));
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
void Axpy<Mem::CUDA>::dv(DT_ * r, const DT_ a, const DT_ * const x, const DT_ * const y, const Index size)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((size)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_axpy<<<grid, block>>>(r, a, x, y, size);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void Axpy<Mem::CUDA>::dv(float *, const float, const float * const, const float * const, const Index);
template void Axpy<Mem::CUDA>::dv(double *, const double, const double * const, const double * const, const Index);

template <typename DT_>
void Axpy<Mem::CUDA>::csr(DT_ * r, const DT_ a, const DT_ * const x, const DT_ * const y, const DT_ * const val, const unsigned long * const col_ind, const unsigned long * const row_ptr, const Index rows, const Index columns, const Index used_elements)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_axpy_mv_csr<<<grid, block>>>(r, a, x, y, val, col_ind, row_ptr, rows);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Axpy<Mem::CUDA>::csr(float *, const float, const float * const, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index);
template void Axpy<Mem::CUDA>::csr(double *, const double, const double * const, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index);

template <typename DT_>
void Axpy<Mem::CUDA>::csr(DT_ * r, const DT_ a, const DT_ * const x, const DT_ * const y, const DT_ * const val, const unsigned int * const col_ind, const unsigned int * const row_ptr, const Index rows, const Index columns, const Index used_elements)
{
  if (r == y)
  {
    hipsparseMatDescr_t descr=0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    DT_ one(1);
    FEAT::LAFEM::Intern::cusparse_axpy_csr(HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind, x, &one, r);

    hipsparseDestroyMatDescr(descr);
  }
  else
  {
    hipMemcpy(r, y, rows * sizeof(DT_), hipMemcpyDeviceToDevice);

    hipsparseMatDescr_t descr=0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    DT_ one(1);
    FEAT::LAFEM::Intern::cusparse_axpy_csr(HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind, x, &one, r);

    hipsparseDestroyMatDescr(descr);
  }

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Axpy<Mem::CUDA>::csr(float *, const float, const float * const, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index);
template void Axpy<Mem::CUDA>::csr(double *, const double, const double * const, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index);

template <typename DT_>
void Axpy<Mem::CUDA>::csrb_intern(DT_ * r, const DT_ a, const DT_ * const x, const DT_ * const y, const DT_ * const val, const unsigned int * const col_ind, const unsigned int * const row_ptr, const Index rows, const Index columns, const Index used_elements, const int blocksize)
{
  if (r == y)
  {
    hipsparseMatDescr_t descr=0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    DT_ one(1);
    FEAT::LAFEM::Intern::cusparse_axpy_csrb(HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind,
        blocksize, x, &one, r);

    hipsparseDestroyMatDescr(descr);
  }
  else
  {
    hipMemcpy(r, y, rows * blocksize * sizeof(DT_), hipMemcpyDeviceToDevice);

    hipsparseMatDescr_t descr=0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    DT_ one(1);
    FEAT::LAFEM::Intern::cusparse_axpy_csrb(HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind,
        blocksize, x, &one, r);

    hipsparseDestroyMatDescr(descr);
  }

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Axpy<Mem::CUDA>::csrb_intern(float *, const float, const float * const, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index, const int);
template void Axpy<Mem::CUDA>::csrb_intern(double *, const double, const double * const, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index, const int);

template <typename DT_, typename IT_>
void Axpy<Mem::CUDA>::ell(DT_ * r, const DT_ a, const DT_ * const x, const DT_ * const y, const DT_ * const val, const IT_ * const col_ind, const IT_ * const cs, const IT_ * const cl, const Index C, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_axpy_mv_ell<<<grid, block>>>(r, a, x, y, val, col_ind, cs, cl, rows, C);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Axpy<Mem::CUDA>::ell(float *, const float, const float * const, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const unsigned int * const, const Index, const Index);
template void Axpy<Mem::CUDA>::ell(double *, const double, const double * const, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const unsigned int * const, const Index, const Index);
template void Axpy<Mem::CUDA>::ell(float *, const float, const float * const, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const unsigned long * const, const Index, const Index);
template void Axpy<Mem::CUDA>::ell(double *, const double, const double * const, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const unsigned long * const, const Index, const Index);

template <typename DT_, typename IT_>
void Axpy<Mem::CUDA>::banded(DT_ * r, const DT_ * const y, const DT_ alpha, const DT_ * const val, const IT_ * const offsets, const DT_ * const x, const Index num_of_offsets, const Index rows, const Index columns)
{
  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_axpy_banded<<<grid, block>>>(r, alpha, x, y, val, offsets, num_of_offsets, rows, columns);
}
template void Axpy<Mem::CUDA>::banded(float *, const float * const, const float, const float * const, const unsigned int * const, const float * const, const Index, const Index, const Index);
template void Axpy<Mem::CUDA>::banded(double *, const double * const, const double, const double * const, const unsigned int * const, const double * const, const Index, const Index, const Index);
template void Axpy<Mem::CUDA>::banded(float *, const float * const, const float, const float * const, const unsigned long * const, const float * const, const Index, const Index, const Index);
template void Axpy<Mem::CUDA>::banded(double *, const double * const, const double, const double * const, const unsigned long * const, const double * const, const Index, const Index, const Index);

template <typename DT_>
void Axpy<Mem::CUDA>::dense(DT_ * r, const DT_ alpha, const DT_ * const y, const DT_ * const val, const DT_ * const x, const Index rows, const Index columns)
{
  DT_ one(1);
  if (r == y)
  {
    FEAT::LAFEM::Intern::cublas_axpy_dense(HIPBLAS_OP_T, (int)rows, (int)columns, &alpha, val, x, &one, r);
  }
  else
  {
    hipMemcpy(r, y, rows * sizeof(DT_), hipMemcpyDeviceToDevice);
    FEAT::LAFEM::Intern::cublas_axpy_dense(HIPBLAS_OP_T, (int)rows, (int)columns, &alpha, val, x, &one, r);
  }

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Axpy<Mem::CUDA>::dense(float * r, const float, const float * const, const float * const, const float * const, const Index, const Index);
template void Axpy<Mem::CUDA>::dense(double * r, const double, const double * const, const double * const, const double * const, const Index, const Index);

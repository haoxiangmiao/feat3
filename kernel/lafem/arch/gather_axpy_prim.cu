#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/gather_axpy_prim.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_gather_axpy_prim_dv_csr(DT_ * b, const DT_* v, const Index* col_ind, const DT_* val, const Index* row_ptr, const DT_ alpha, const Index size, const Index offset)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= size)
          return;

        // loop over all gather-matrix rows
        DT_ sum(DT_(0));
        for (Index i(row_ptr[idx]) ; i < row_ptr[idx + 1] ; ++i)
        {
          sum += DT_(val[i]) * DT_(v[col_ind[i]]);
        }
        b[offset + idx] += alpha * sum;
      }
    }
  }
}

using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
void GatherAxpyPrim<Mem::CUDA>::dv_csr(DT_ * v, const DT_* b, const Index* col_ind, const DT_* val, const Index* row_ptr, const DT_ alpha, const Index size, const Index offset)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_spmv;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((size)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_gather_axpy_prim_dv_csr<<<grid, block>>>(v, b, col_ind, val, row_ptr, alpha, size, offset);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void GatherAxpyPrim<Mem::CUDA>::dv_csr(float *, const float*, const Index*, const float*, const Index*, const float alpha, const Index, const Index);
template void GatherAxpyPrim<Mem::CUDA>::dv_csr(double *, const double*, const Index*, const double*, const Index*, const double alpha, const Index, const Index);

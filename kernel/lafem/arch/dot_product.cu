// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/dot_product.hpp>
#include <kernel/lafem/arch/component_product.hpp>
#include <kernel/util/exception.hpp>

// includes, CUDA
#include <hipblas.h>

namespace FEAST
{
  namespace Util
  {
    namespace Intern
    {
      extern hipblasHandle_t cublas_handle;
    }
  }

  namespace LAFEM
  {
    namespace Intern
    {

      float cuda_dot_product(const float * x, const float * y, const Index size)
      {
        float result;
        if (HIPBLAS_STATUS_SUCCESS != hipblasSdot(Util::Intern::cublas_handle, size, x, 1, y, 1, &result))
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasSdot failed!");
        hipDeviceSynchronize();
        return result;
      }

      double cuda_dot_product(const double * x, const double * y, const Index size)
      {
        double result;
        if (HIPBLAS_STATUS_SUCCESS != hipblasDdot(Util::Intern::cublas_handle, size, x, 1, y, 1, &result))
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasDdot failed!");
        hipDeviceSynchronize();
        return result;
      }
    }
  }
}

using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
DT_ DotProduct<Mem::CUDA, Algo::CUDA>::value(const DT_ * const x, const DT_ * const y, const Index size)
{
  DT_ result = Intern::cuda_dot_product(x, y, size);
  return result;
}

template float DotProduct<Mem::CUDA, Algo::CUDA>::value(const float * const, const float * const, const Index);
template double DotProduct<Mem::CUDA, Algo::CUDA>::value(const double * const, const double * const, const Index);

template <typename DT_>
DT_ TripleDotProduct<Mem::CUDA, Algo::CUDA>::value(const DT_ * const x, const DT_ * const y, const DT_ * const z, const Index size)
{
  DT_ * temp;
  hipMalloc((void **) &temp, size * sizeof(DT_));
  ComponentProduct<Mem::CUDA, Algo::CUDA>::value(temp, y, z, size);
  DT_ result = Intern::cuda_dot_product(x, temp, size);
  hipFree(temp);
  return result;
}

template float TripleDotProduct<Mem::CUDA, Algo::CUDA>::value(const float * const x, const float * const y, const float * const z, const Index size);
template double TripleDotProduct<Mem::CUDA, Algo::CUDA>::value(const double * const x, const double * const y, const double * const z, const Index size);

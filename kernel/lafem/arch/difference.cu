#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/difference.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_difference(DT_ * r, const DT_ * x, const DT_ * y, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        r[idx] = x[idx] - y[idx];
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
void Difference<Mem::CUDA>::value(DT_ * r, const DT_ * const x, const DT_ * const y, const Index size)
{
  Index blocksize = Util::MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((size)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_difference<<<grid, block>>>(r, x, y, size);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void Difference<Mem::CUDA>::value(float *, const float * const, const float * const, const Index);
template void Difference<Mem::CUDA>::value(double *, const double * const, const double * const, const Index);

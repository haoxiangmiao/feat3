#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/norm.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

// includes, CUDA
#include <hipblas.h>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      float cuda_norm2(const float * x, const Index size)
      {
        float result;
        hipblasStatus_t status;
        status = hipblasSnrm2(Util::Intern::cublas_handle, size, x, 1, &result);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cublasnrm2 failed with status code: " + stringify(status));
        hipDeviceSynchronize();
        return result;
      }

      double cuda_norm2(const double * x, const Index size)
      {
        double result;
        hipblasStatus_t status;
        status = hipblasDnrm2(Util::Intern::cublas_handle, size, x, 1, &result);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cublasnrm2 failed with status code: " + stringify(status));
        hipDeviceSynchronize();
        return result;
      }
    }
  }
}

using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
DT_ Norm2<Mem::CUDA>::value(const DT_ * const x, const Index size)
{
  DT_ result = Intern::cuda_norm2(x, size);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
  return result;
}

template float Norm2<Mem::CUDA>::value(const float * const, const Index);
template double Norm2<Mem::CUDA>::value(const double * const, const Index);

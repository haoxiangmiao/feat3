#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/norm.hpp>
#include <kernel/util/exception.hpp>

// includes, CUDA
#include <hipblas.h>

namespace FEAST
{
  namespace Util
  {
    namespace Intern
    {
      extern hipblasHandle_t cublas_handle;
    }
  }

  namespace LAFEM
  {
    namespace Intern
    {
      float cuda_norm2(const float * x, const Index size)
      {
        float result;
        if (HIPBLAS_STATUS_SUCCESS != hipblasSnrm2(Util::Intern::cublas_handle, size, x, 1, &result))
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasSnrm2 failed!");
        hipDeviceSynchronize();
        return result;
      }

      double cuda_norm2(const double * x, const Index size)
      {
        double result;
        if (HIPBLAS_STATUS_SUCCESS != hipblasDnrm2(Util::Intern::cublas_handle, size, x, 1, &result))
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasDnrm2 failed!");
        hipDeviceSynchronize();
        return result;
      }
    }
  }
}

using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
DT_ Norm2<Mem::CUDA, Algo::CUDA>::value(const DT_ * const x, const Index size)
{
  DT_ result = Intern::cuda_norm2(x, size);
  return result;
}

template float Norm2<Mem::CUDA, Algo::CUDA>::value(const float * const, const Index);
template double Norm2<Mem::CUDA, Algo::CUDA>::value(const double * const, const Index);

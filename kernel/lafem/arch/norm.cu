#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/norm.hpp>
#include <kernel/util/exception.hpp>

// includes, CUDA
#include <hipblas.h>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      extern hipblasHandle_t cublas_handle;

      float cuda_norm2(const float * x, const Index size)
      {
        float result;
        hipblasSnrm2(cublas_handle, size, x, 1, &result);
        return result;
      }

      double cuda_norm2(const double * x, const Index size)
      {
        double result;
        hipblasDnrm2(cublas_handle, size, x, 1, &result);
        return result;
      }
    }
  }
}

using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
DT_ Norm2<Mem::CUDA, Algo::CUDA>::value(const DT_ * const x, const Index size)
{
  DT_ result = Intern::cuda_norm2(x, size);
  return result;
}

template float Norm2<Mem::CUDA, Algo::CUDA>::value(const float * const, const Index);
template double Norm2<Mem::CUDA, Algo::CUDA>::value(const double * const, const Index);

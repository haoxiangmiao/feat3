#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/component_invert.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_component_invert(DT_ * r, const DT_ * x, const DT_ s, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        r[idx] = s / x[idx];
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
void ComponentInvert<Mem::CUDA>::value(DT_ * r, const DT_ * const x, const DT_ s, const Index size)
{
  Index blocksize = Util::MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((size)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_component_invert<<<grid, block>>>(r, x, s, size);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void ComponentInvert<Mem::CUDA>::value(float *, const float * const, const float, const Index);
template void ComponentInvert<Mem::CUDA>::value(double *, const double * const, const double, const Index);

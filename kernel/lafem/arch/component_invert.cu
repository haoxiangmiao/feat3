#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/component_invert.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_component_invert(DT_ * r, const DT_ * x, const DT_ s, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        r[idx] = s / x[idx];
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_>
void ComponentInvert<Mem::CUDA, Algo::CUDA>::value(DT_ * r, const DT_ * const x, const DT_ s, const Index size)
{
  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((size)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_component_invert<<<grid, block>>>(r, x, s, size);
}

template void ComponentInvert<Mem::CUDA, Algo::CUDA>::value(float *, const float * const, const float, const Index);
template void ComponentInvert<Mem::CUDA, Algo::CUDA>::value(double *, const double * const, const double, const Index);

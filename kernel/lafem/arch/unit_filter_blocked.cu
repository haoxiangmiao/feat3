#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/unit_filter_blocked.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

/// \cond internal
namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_unit_filter_blocked_rhs(DT_ * v, const DT_ * sv_elements, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;

        Index block_size = Index(BlockSize_);
        for(Index j(0) ; j < block_size; ++j)
          v[block_size* sv_indices[idx] + j] = sv_elements[block_size * idx + j];
      }

      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_unit_filter_blocked_def(DT_ * v, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;

        Index block_size = Index(BlockSize_);
        for(Index j(0) ; j < block_size; ++j)
          v[block_size * sv_indices[idx] + j] = DT_(0);
      }
    }
  }
}


using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_, typename IT_, int BlockSize_>
void UnitFilterBlocked<Mem::CUDA>::filter_rhs(DT_ * v, const DT_ * const sv_elements, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = Util::MemoryPool<Mem::CUDA>::blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_unit_filter_blocked_rhs<DT_, IT_, BlockSize_><<<grid, block>>>(v, sv_elements, sv_indices, ue);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned long, 1>(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned long, 1>(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned int, 1>(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned int, 1>(double *, const double * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned long, 2>(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned long, 2>(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned int, 2>(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned int, 2>(double *, const double * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned long, 3>(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned long, 3>(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned int, 3>(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned int, 3>(double *, const double * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned long, 4>(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned long, 4>(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned int, 4>(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned int, 4>(double *, const double * const, const unsigned int * const, const Index);

template <typename DT_, typename IT_, int BlockSize_>
void UnitFilterBlocked<Mem::CUDA>::filter_def(DT_ * v, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = Util::MemoryPool<Mem::CUDA>::blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_unit_filter_blocked_def<DT_, IT_, BlockSize_><<<grid, block>>>(v, sv_indices, ue);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned long, 1>(float *, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned long, 1>(double *,  const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned int, 1>(float *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned int, 1>(double *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned long, 2>(float *, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned long, 2>(double *,  const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned int, 2>(float *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned int, 2>(double *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned long, 3>(float *, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned long, 3>(double *,  const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned int, 3>(float *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned int, 3>(double *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned long, 4>(float *, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned long, 4>(double *,  const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned int, 4>(float *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned int, 4>(double *,  const unsigned int * const, const Index);

/// \endcond

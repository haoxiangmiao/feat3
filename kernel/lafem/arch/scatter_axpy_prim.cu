#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/scatter_axpy_prim.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

namespace FEAST
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_, typename IT_>
      __global__ void cuda_scatter_axpy_prim_dv_csr(DT_ * v, const DT_* b, const IT_* col_ind, const DT_* val, const IT_* row_ptr, const DT_ alpha, const Index size)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= size)
          return;

          // skip empty rows
          if(row_ptr[idx] == row_ptr[idx + 1])
            return;

          DT_ sum(0);
          for (Index i(row_ptr[idx]) ; i < row_ptr[idx + 1] ; ++i)
          {
            sum += val[i] * (b[col_ind[i]]);
          }
          v[idx] += alpha * sum;
      }
    }
  }
}

using namespace FEAST;
using namespace FEAST::LAFEM;
using namespace FEAST::LAFEM::Arch;

template <typename DT_, typename IT_>
void ScatterAxpyPrim<Mem::CUDA>::dv_csr(DT_ * v, const DT_* b, const IT_* col_ind, const DT_* val, const IT_* row_ptr, const DT_ alpha, const Index size)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_spmv;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((size)/(double)(block.x));

  FEAST::LAFEM::Intern::cuda_scatter_axpy_prim_dv_csr<<<grid, block>>>(v, b, col_ind, val, row_ptr, alpha, size);
#ifdef FEAST_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void ScatterAxpyPrim<Mem::CUDA>::dv_csr(float *, const float*, const unsigned long*, const float*, const unsigned long*, const float alpha, const Index);
template void ScatterAxpyPrim<Mem::CUDA>::dv_csr(double *, const double*, const unsigned long*, const double*, const unsigned long*, const double alpha, const Index);
template void ScatterAxpyPrim<Mem::CUDA>::dv_csr(float *, const float*, const unsigned int*, const float*, const unsigned int*, const float alpha, const Index);
template void ScatterAxpyPrim<Mem::CUDA>::dv_csr(double *, const double*, const unsigned int*, const double*, const unsigned int*, const double alpha, const Index);

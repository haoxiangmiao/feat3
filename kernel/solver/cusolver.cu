// includes, FEAST
#include <kernel/base_header.hpp>

#ifdef FEAST_HAVE_CUSOLVER
#include <kernel/archs.hpp>
#include <kernel/util/exception.hpp>

#include <hipsolver.h>
#include "hipsparse.h"

using namespace FEAST;

namespace FEAST
{
  namespace Solver
  {
    namespace Intern
    {
      int cuda_lu(int n, int nnzA, const double * csrValA, const int * csrRowPtrA, const int * csrColIndA,
          const double * b, double * x)
      {
        hipsolverSpHandle_t handle;
        hipsolverSpCreate(&handle);

        hipsparseMatDescr_t descr;
        hipsparseCreateMatDescr(&descr);
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        int singularity;
        hipsolverStatus_t status = cusolverSpDcsrlsvluHost(handle, n, nnzA, descr, csrValA, csrRowPtrA, csrColIndA, b, 0.0, 1, x, &singularity);

        hipsparseDestroyMatDescr(descr);
        hipsolverSpDestroy(handle);

#ifdef FEAST_DEBUG_MODE
        hipDeviceSynchronize();
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return (status != HIPSOLVER_STATUS_SUCCESS);
      }


      int cuda_qr(int m, int nnz, const double * csrValA, const int * csrRowPtrA, const int * csrColIndA,
          const double * b, double * x)
      {
        hipsolverSpHandle_t handle;
        hipsolverSpCreate(&handle);

        hipsparseMatDescr_t descr;
        hipsparseCreateMatDescr(&descr);
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        int singularity;
        hipsolverStatus_t status = hipsolverSpDcsrlsvqr(handle, m, nnz, descr, csrValA, csrRowPtrA, csrColIndA, b, 0.0, 1, x, &singularity);

        hipsparseDestroyMatDescr(descr);
        hipsolverSpDestroy(handle);
#ifdef FEAST_DEBUG_MODE
        hipDeviceSynchronize();
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return (status != HIPSOLVER_STATUS_SUCCESS);
      }
    }
  }
}
#endif // FEAST_HAVE_CUSOLVER

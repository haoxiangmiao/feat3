#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>

#include <kernel/archs.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

#include "hipsparse.h"

// http://docs.nvidia.com/cuda/cusparse/#cusparse-lt-t-gt-csrilu02_solve


using namespace FEAST;

namespace FEAST
{
  namespace Solver
  {
    namespace Intern
    {
      // CSR
      struct CudaIluSolveInfo
      {
        hipsparseMatDescr_t descr_M;
        hipsparseMatDescr_t descr_L;
        hipsparseMatDescr_t descr_U;
        csrilu02Info_t info_M;
        csrsv2Info_t info_L;
        csrsv2Info_t info_U;
        hipsparseOperation_t trans_L;
        hipsparseOperation_t trans_U;
        hipsparseSolvePolicy_t policy_M;
        hipsparseSolvePolicy_t policy_L;
        hipsparseSolvePolicy_t policy_U;
        void * pBuffer;
        double * z;
        int m;
        int nnz;
      };

      void * cuda_ilu_init_symbolic(int m, int nnz, double * csrVal, int * csrRowPtr, int * csrColInd)
      {
        double * z;
        hipMalloc((void**)&z, m * sizeof(double));

        hipsparseMatDescr_t descr_M = 0;
        hipsparseMatDescr_t descr_L = 0;
        hipsparseMatDescr_t descr_U = 0;
        csrilu02Info_t info_M  = 0;
        csrsv2Info_t  info_L  = 0;
        csrsv2Info_t  info_U  = 0;
        int pBufferSize_M;
        int pBufferSize_L;
        int pBufferSize_U;
        int pBufferSize;
        void *pBuffer = 0;
        int structural_zero;
        const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        hipsparseCreateMatDescr(&descr_M);
        hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

        hipsparseCreateMatDescr(&descr_L);
        hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

        hipsparseCreateMatDescr(&descr_U);
        hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

        hipsparseCreateCsrilu02Info(&info_M);
        hipsparseCreateCsrsv2Info(&info_L);
        hipsparseCreateCsrsv2Info(&info_U);

        hipsparseDcsrilu02_bufferSize(Util::Intern::cusparse_handle, m, nnz,
                descr_M, csrVal, csrRowPtr, csrColInd, info_M, &pBufferSize_M);
        hipsparseDcsrsv2_bufferSize(Util::Intern::cusparse_handle, trans_L, m, nnz,
                descr_L, csrVal, csrRowPtr, csrColInd, info_L, &pBufferSize_L);
        hipsparseDcsrsv2_bufferSize(Util::Intern::cusparse_handle, trans_U, m, nnz,
                descr_U, csrVal, csrRowPtr, csrColInd, info_U, &pBufferSize_U);

        pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

        hipMalloc((void**)&pBuffer, pBufferSize);

        hipsparseDcsrilu02_analysis(Util::Intern::cusparse_handle, m, nnz, descr_M,
                csrVal, csrRowPtr, csrColInd, info_M,
                    policy_M, pBuffer);
        hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(Util::Intern::cusparse_handle, info_M, &structural_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }

        hipsparseDcsrsv2_analysis(Util::Intern::cusparse_handle, trans_L, m, nnz, descr_L,
                csrVal, csrRowPtr, csrColInd, info_L, policy_L, pBuffer);

        hipsparseDcsrsv2_analysis(Util::Intern::cusparse_handle, trans_U, m, nnz, descr_U,
                csrVal, csrRowPtr, csrColInd, info_U, policy_U, pBuffer);


#ifdef FEAST_DEBUG_MODE
        hipDeviceSynchronize();
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        CudaIluSolveInfo * info = new CudaIluSolveInfo;
        info->descr_M = descr_M;
        info->descr_L = descr_L;
        info->descr_U = descr_U;
        info->info_M  = info_M;
        info->info_L  = info_L;
        info->info_U  = info_U;
        info->trans_L = trans_L;
        info->trans_U = trans_U;
        info->policy_M = policy_M;
        info->policy_L = policy_L;
        info->policy_U = policy_U;
        info->pBuffer = pBuffer;
        info->z = z;
        info->m = m;
        info->nnz = nnz;

        return (void*)info;
      }

      void cuda_ilu_init_numeric(double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;

        hipsparseDcsrilu02(Util::Intern::cusparse_handle, info->m, info->nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->info_M, info->policy_M, info->pBuffer);
        int numerical_zero;
        hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &numerical_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }
      }

      int cuda_ilu_apply(double * y, const double * x, double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;
        const double alpha = 1.;

        hipsparseDcsrsv2_solve(Util::Intern::cusparse_handle, info->trans_L, info->m, info->nnz, &alpha, info->descr_L,
               csrVal, csrRowPtr, csrColInd, info->info_L,
                  x, info->z, info->policy_L, info->pBuffer);

        hipsparseDcsrsv2_solve(Util::Intern::cusparse_handle, info->trans_U, info->m, info->nnz, &alpha, info->descr_U,
               csrVal, csrRowPtr, csrColInd, info->info_U,
                  info->z, y, info->policy_U, info->pBuffer);

#ifdef FEAST_DEBUG_MODE
        hipDeviceSynchronize();
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return 0;
      }

      void cuda_ilu_done(void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;

        hipFree(info->z);
        hipFree(info->pBuffer);
        hipsparseDestroyMatDescr(info->descr_M);
        hipsparseDestroyMatDescr(info->descr_L);
        hipsparseDestroyMatDescr(info->descr_U);
        hipsparseDestroyCsrilu02Info(info->info_M);
        hipsparseDestroyCsrsv2Info(info->info_L);
        hipsparseDestroyCsrsv2Info(info->info_U);

        delete info;
      }

      // BCSR
      struct CudaIluBSolveInfo
      {
        hipsparseMatDescr_t descr_M;
        hipsparseMatDescr_t descr_L;
        hipsparseMatDescr_t descr_U;
        bsrilu02Info_t info_M;
        bsrsv2Info_t info_L;
        bsrsv2Info_t info_U;
        hipsparseOperation_t trans_L;
        hipsparseOperation_t trans_U;
        hipsparseDirection_t dir;
        hipsparseSolvePolicy_t policy_M;
        hipsparseSolvePolicy_t policy_L;
        hipsparseSolvePolicy_t policy_U;
        void * pBuffer;
        double * z;
        int m;
        int nnz;
        int blocksize;
      };

      void * cuda_ilub_init_symbolic(int m, int nnz, double * csrVal, int * csrRowPtr, int * csrColInd, const int blocksize)
      {
        double * z;
        hipMalloc((void**)&z, m * blocksize * sizeof(double));

        hipsparseMatDescr_t descr_M = 0;
        hipsparseMatDescr_t descr_L = 0;
        hipsparseMatDescr_t descr_U = 0;
        bsrilu02Info_t info_M  = 0;
        bsrsv2Info_t  info_L  = 0;
        bsrsv2Info_t  info_U  = 0;
        int pBufferSize_M;
        int pBufferSize_L;
        int pBufferSize_U;
        int pBufferSize;
        void *pBuffer = 0;
        int structural_zero;
        const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;

        hipsparseCreateMatDescr(&descr_M);
        hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

        hipsparseCreateMatDescr(&descr_L);
        hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

        hipsparseCreateMatDescr(&descr_U);
        hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

        hipsparseCreateBsrilu02Info(&info_M);
        hipsparseCreateBsrsv2Info(&info_L);
        hipsparseCreateBsrsv2Info(&info_U);

        hipsparseDbsrilu02_bufferSize(Util::Intern::cusparse_handle, dir, m, nnz,
                descr_M, csrVal, csrRowPtr, csrColInd, blocksize, info_M, &pBufferSize_M);
        hipsparseDbsrsv2_bufferSize(Util::Intern::cusparse_handle, dir, trans_L, m, nnz,
                descr_L, csrVal, csrRowPtr, csrColInd, blocksize, info_L, &pBufferSize_L);
        hipsparseDbsrsv2_bufferSize(Util::Intern::cusparse_handle, dir, trans_U, m, nnz,
                descr_U, csrVal, csrRowPtr, csrColInd, blocksize, info_U, &pBufferSize_U);

        pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

        hipMalloc((void**)&pBuffer, pBufferSize);

        hipsparseDbsrilu02_analysis(Util::Intern::cusparse_handle, dir, m, nnz, descr_M,
                csrVal, csrRowPtr, csrColInd, blocksize, info_M,
                    policy_M, pBuffer);
        hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(Util::Intern::cusparse_handle, info_M, &structural_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }

        hipsparseDbsrsv2_analysis(Util::Intern::cusparse_handle, dir, trans_L, m, nnz, descr_L,
                csrVal, csrRowPtr, csrColInd, blocksize, info_L, policy_L, pBuffer);

        hipsparseDbsrsv2_analysis(Util::Intern::cusparse_handle, dir, trans_U, m, nnz, descr_U,
                csrVal, csrRowPtr, csrColInd, blocksize, info_U, policy_U, pBuffer);


#ifdef FEAST_DEBUG_MODE
        hipDeviceSynchronize();
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        CudaIluBSolveInfo * info = new CudaIluBSolveInfo;
        info->descr_M = descr_M;
        info->descr_L = descr_L;
        info->descr_U = descr_U;
        info->info_M  = info_M;
        info->info_L  = info_L;
        info->info_U  = info_U;
        info->trans_L = trans_L;
        info->trans_U = trans_U;
        info->dir = dir;
        info->policy_M = policy_M;
        info->policy_L = policy_L;
        info->policy_U = policy_U;
        info->pBuffer = pBuffer;
        info->z = z;
        info->m = m;
        info->nnz = nnz;
        info->blocksize = blocksize;

        return (void*)info;
      }

      void cuda_ilub_init_numeric(double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;

        hipsparseDbsrilu02(Util::Intern::cusparse_handle, info->dir, info->m, info->nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_M, info->policy_M, info->pBuffer);
        int numerical_zero;
        hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &numerical_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }
      }

      int cuda_ilub_apply(double * y, const double * x, double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;
        const double alpha = 1.;

        hipsparseDbsrsv2_solve(Util::Intern::cusparse_handle, info->dir, info->trans_L, info->m, info->nnz, &alpha, info->descr_L,
               csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_L,
                  x, info->z, info->policy_L, info->pBuffer);

        hipsparseDbsrsv2_solve(Util::Intern::cusparse_handle, info->dir, info->trans_U, info->m, info->nnz, &alpha, info->descr_U,
               csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_U,
                  info->z, y, info->policy_U, info->pBuffer);

#ifdef FEAST_DEBUG_MODE
        hipDeviceSynchronize();
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return 0;
      }

      void cuda_ilub_done(void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;

        hipFree(info->z);
        hipFree(info->pBuffer);
        hipsparseDestroyMatDescr(info->descr_M);
        hipsparseDestroyMatDescr(info->descr_L);
        hipsparseDestroyMatDescr(info->descr_U);
        hipsparseDestroyBsrilu02Info(info->info_M);
        hipsparseDestroyBsrsv2Info(info->info_L);
        hipsparseDestroyBsrsv2Info(info->info_U);

        delete info;
      }
    }
  }
}

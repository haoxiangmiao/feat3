#include "hip/hip_runtime.h"
// includes, FEAST
#include <kernel/base_header.hpp>

#include <kernel/archs.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

#include "hipsparse.h"

// http://docs.nvidia.com/cuda/cusparse/#cusparse-lt-t-gt-csrilu02_solve


using namespace FEAST;

namespace FEAST
{
  namespace Util
  {
    namespace Intern
    {
      extern hipsparseHandle_t cusparse_handle;
    }
  }

  namespace Solver
  {
    namespace Intern
    {

      struct CudaIluSolveInfo
      {
        hipsparseMatDescr_t descr_M;
        hipsparseMatDescr_t descr_L;
        hipsparseMatDescr_t descr_U;
        csrilu02Info_t info_M;
        csrsv2Info_t info_L;
        csrsv2Info_t info_U;
        hipsparseOperation_t trans_L;
        hipsparseOperation_t trans_U;
        hipsparseSolvePolicy_t policy_M;
        hipsparseSolvePolicy_t policy_L;
        hipsparseSolvePolicy_t policy_U;
        void * pBuffer;
        double * z;
        int m;
        int nnz;
      };

      void * cuda_ilu_init_symbolic(int m, int nnz, double * csrVal, int * csrRowPtr, int * csrColInd)
      {
        double * z;
        hipMalloc((void**)&z, m * sizeof(double));

        hipsparseMatDescr_t descr_M = 0;
        hipsparseMatDescr_t descr_L = 0;
        hipsparseMatDescr_t descr_U = 0;
        csrilu02Info_t info_M  = 0;
        csrsv2Info_t  info_L  = 0;
        csrsv2Info_t  info_U  = 0;
        int pBufferSize_M;
        int pBufferSize_L;
        int pBufferSize_U;
        int pBufferSize;
        void *pBuffer = 0;
        int structural_zero;
        const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        hipsparseCreateMatDescr(&descr_M);
        hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

        hipsparseCreateMatDescr(&descr_L);
        hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

        hipsparseCreateMatDescr(&descr_U);
        hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

        hipsparseCreateCsrilu02Info(&info_M);
        hipsparseCreateCsrsv2Info(&info_L);
        hipsparseCreateCsrsv2Info(&info_U);

        hipsparseDcsrilu02_bufferSize(Util::Intern::cusparse_handle, m, nnz,
                descr_M, csrVal, csrRowPtr, csrColInd, info_M, &pBufferSize_M);
        hipsparseDcsrsv2_bufferSize(Util::Intern::cusparse_handle, trans_L, m, nnz,
                descr_L, csrVal, csrRowPtr, csrColInd, info_L, &pBufferSize_L);
        hipsparseDcsrsv2_bufferSize(Util::Intern::cusparse_handle, trans_U, m, nnz,
                descr_U, csrVal, csrRowPtr, csrColInd, info_U, &pBufferSize_U);

        pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

        hipMalloc((void**)&pBuffer, pBufferSize);

        hipsparseDcsrilu02_analysis(Util::Intern::cusparse_handle, m, nnz, descr_M,
                csrVal, csrRowPtr, csrColInd, info_M,
                    policy_M, pBuffer);
        hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(Util::Intern::cusparse_handle, info_M, &structural_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }

        hipsparseDcsrsv2_analysis(Util::Intern::cusparse_handle, trans_L, m, nnz, descr_L,
                csrVal, csrRowPtr, csrColInd, info_L, policy_L, pBuffer);

        hipsparseDcsrsv2_analysis(Util::Intern::cusparse_handle, trans_U, m, nnz, descr_U,
                csrVal, csrRowPtr, csrColInd, info_U, policy_U, pBuffer);


#ifdef FEAST_DEBUG_MODE
        hipDeviceSynchronize();
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        CudaIluSolveInfo * info = new CudaIluSolveInfo;
        info->descr_M = descr_M;
        info->descr_L = descr_L;
        info->descr_U = descr_U;
        info->info_M  = info_M;
        info->info_L  = info_L;
        info->info_U  = info_U;
        info->trans_L = trans_L;
        info->trans_U = trans_U;
        info->policy_M = policy_M;
        info->policy_L = policy_L;
        info->policy_U = policy_U;
        info->pBuffer = pBuffer;
        info->z = z;
        info->m = m;
        info->nnz = nnz;

        return (void*)info;
      }

      void cuda_ilu_init_numeric(double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;

        hipsparseDcsrilu02(Util::Intern::cusparse_handle, info->m, info->nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->info_M, info->policy_M, info->pBuffer);
        int numerical_zero;
        hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &numerical_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }
      }

      int cuda_ilu_apply(double * y, const double * x, double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;
        const double alpha = 1.;

        hipsparseDcsrsv2_solve(Util::Intern::cusparse_handle, info->trans_L, info->m, info->nnz, &alpha, info->descr_L,
               csrVal, csrRowPtr, csrColInd, info->info_L,
                  x, info->z, info->policy_L, info->pBuffer);

        hipsparseDcsrsv2_solve(Util::Intern::cusparse_handle, info->trans_U, info->m, info->nnz, &alpha, info->descr_U,
               csrVal, csrRowPtr, csrColInd, info->info_U,
                  info->z, y, info->policy_U, info->pBuffer);

#ifdef FEAST_DEBUG_MODE
        hipDeviceSynchronize();
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return 0;
      }

      void cuda_ilu_done(void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;

        hipFree(info->z);
        hipFree(info->pBuffer);
        hipsparseDestroyMatDescr(info->descr_M);
        hipsparseDestroyMatDescr(info->descr_L);
        hipsparseDestroyMatDescr(info->descr_U);
        hipsparseDestroyCsrilu02Info(info->info_M);
        hipsparseDestroyCsrsv2Info(info->info_L);
        hipsparseDestroyCsrsv2Info(info->info_U);

        delete info;
      }
    }
  }
}

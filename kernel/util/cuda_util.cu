// includes, FEAST
#include <kernel/util/cuda_util.hpp>
#include <kernel/util/string.hpp>
#include <kernel/util/exception.hpp>


void FEAST::Util::cuda_set_device(const int device)
{
  hipSetDevice(device);
}

void * FEAST::Util::cuda_malloc_host(const Index bytes)
{
  void * memory(nullptr);
  if (bytes == 0)
    return memory;

  if (hipErrorOutOfMemory == hipHostMalloc((void**)&memory, bytes, hipHostMallocMapped))
    throw InternalError(__func__, __FILE__, __LINE__, "MemoryPool<CUDA> cuda pinned allocation error (hipErrorOutOfMemory)");
  if (memory == nullptr)
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_malloc_host allocation error (null pointer returned)");
  return memory;
}

void FEAST::Util::cuda_free_host(void * address)
{
  if (address == nullptr)
    return;

  if (hipSuccess != hipHostFree(address))
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_free_host: hipHostFree failed!");
}

void FEAST::Util::cuda_check_last_error()
{
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occured in execution!\n" + stringify(hipGetErrorString(last_error)));
}

void * FEAST::Util::cuda_get_device_pointer(void * host)
{
  void * device(nullptr);
  if (hipSuccess != hipHostGetDevicePointer((void**)&device, host, 0))
    throw InternalError(__func__, __FILE__, __LINE__, "hipHostGetDevicePointer failed!");
  return device;
}
